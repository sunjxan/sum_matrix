#include "hip/hip_runtime.h"
#include "common.hpp"

// 朴素实现，注意iy和ix对行列的编码

__global__ void kernel(const real (*A)[N], const real (*B)[N], real (*C)[N])
{
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (iy < M && ix < N) {
        C[iy][ix] = C[iy][ix] = A[iy][ix] + B[iy][ix];
    }
}

void sum_matrix(const real *A, const real *B, real *C)
{
    const real (*nA)[N] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    dim3 block_size(32, 32);
    // N是列对应x，M是行对应y
    dim3 grid_size(DIVUP(N, block_size.x), DIVUP(M, block_size.y));
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}
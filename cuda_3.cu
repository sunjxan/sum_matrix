#include "hip/hip_runtime.h"
#include "common.hpp"

// 朴素实现，注意ix和iy对行列的编码

__global__ void kernel(const real (*A)[N], const real (*B)[N], real (*C)[N])
{
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (iy < N && ix < M) {
        C[ix][iy] = __ldg(&A[ix][iy]) + __ldg(&B[ix][iy]);
    }
}

void sum_matrix(const real *A, const real *B, real *C)
{
    const real (*nA)[N] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    dim3 block_size(32, 32);
    // N是列对应y，M是行对应x
    dim3 grid_size(DIVUP(M, block_size.x), DIVUP(N, block_size.y));
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}
